#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <hip/hip_complex.h>
#include "kernel.h"
#define _USE_MATH_DEFINES
#include <math.h>
#include <time.h>
#include ""

const int SIZE = 1024;
const double MINFREQ = 1e-6;
using namespace std;
bool debug;

__global__ void VectorAdd(double *freq, hipComplex *Ahat, hipComplex *data, hipComplex *Poles, frequency *frequencyInfo, int *Apattern, int Ahat_size, int NComplexPoles, int NRealPoles)
{
	double real=0;
	double imag=0;
	double denum=0;
	int poleNumb = 0;
	int NRow = (*frequencyInfo).fpointCount;
	double s;
	int test = 0;
		int col = blockIdx.x;
		int row = threadIdx.x;
		if (col > 17) {
			poleNumb = col - 17;
		}
		else {
			poleNumb = col;
		}

		real = Poles[poleNumb].x;
		imag = Poles[poleNumb].y;

			s = 2 * M_PI*freq[row];

		//real		
		if (Apattern[col] == 1) {
			Ahat[col*NRow + row].x = -real / (pow(real, 2) + pow(s, 2));
			Ahat[col*NRow + row].y = -s / (pow(real, 2) + pow(s, 2));
		}
		else if (Apattern[col] == 2) {
			denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
			Ahat[col*NRow + row].x = -2 * (real*(pow(real, 2) + pow(s, 2) + pow(imag, 2))) / denum;
			Ahat[col*NRow + row].y = -2 * (s *(pow(real, 2) + pow(s, 2) - pow(imag, 2))) / denum;
		}
		else if (Apattern[col] == 3) {
			denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
			Ahat[(col)*NRow + row].x = (-2 * imag*(pow(real, 2) - pow(s, 2) + pow(imag, 2))) / denum;
			Ahat[(col)*NRow + row].y = (-4 * real*imag*s) / denum;
		}
		else if (Apattern[col] == 4) {
			Ahat[col*NRow + row].x = 1;
			Ahat[col*NRow + row].y = 0;
		}
		else if (Apattern[col] == -1) {
			denum = pow(real, 2) + pow(imag, 2) - 2 * imag*s + pow(s, 2);
			Ahat[col*NRow + row].x = (real*data[row].x - data[row].y*s + data[row].y*imag) / denum;
			Ahat[col*NRow + row].y = (s*data[row].x - data[row].x*imag + data[row].y*real) / denum;
		}
		else if (Apattern[col] == -2) {
			denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
			Ahat[col*NRow + row].x = 2 * (pow(real, 3)*data[row].x - pow(real, 2)*data[row].y*s + real* data[row].x*pow(s, 2) + real* data[row].x*pow(imag, 2) + s*data[row].y*pow(imag, 2) - data[row].y*pow(s, 3)) / denum;
			Ahat[col*NRow + row].y = 2 * (pow(real, 3)*data[row].y + pow(real, 2)*data[row].x*s + real* data[row].y*pow(s, 2) + real* data[row].y*pow(imag, 2) - s*data[row].x*pow(imag, 2) + data[row].x*pow(s, 3)) / denum;
		}
		else if (Apattern[col] == -3) {
			denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
			Ahat[col*NRow + row].x = 2 * imag*(pow(real, 2)*data[row].x - data[row].x* pow(s, 2) + data[row].x*pow(imag, 2) - 2 * real*data[row].y*s) / denum;
			Ahat[col*NRow + row].y = 2 * imag*(2 * data[row].x*real*s + pow(real, 2)*data[row].y - data[row].y*pow(s, 2) + data[row].y*pow(imag, 2)) / denum;
		}

}
//
//void readFile(char *fileName, int *data)
//{
//	int i = threadIdx.x;
//	if (i < n)
//		c[i] = a[i] + b[i];
//}

int main()
{
	debug = true;
	string dataFileName = "radial_stub^S.txt";
	int NRealPoles = 1;
	int NComplexPoles = 16;

	//###########################Reading File########################################



	//Ali: var to store freq points in data (upto 1024 data points)
	double *freq;
	hipMallocManaged(&freq, SIZE * sizeof(double));

	//Ali: store collected data in complex form (upto 1024 data points)
	hipComplex *data;
	hipMallocManaged(&data, SIZE * sizeof(hipComplex));

	//Ali: store info about the stroed data
	//		-lowest freq
	//		-highest freq
	//		-number of freq points
	frequency *frequencyInfo;
	hipMallocManaged(&frequencyInfo, sizeof(frequency));

	std::string::size_type sz;


	ifstream infile(dataFileName);

	//Ali: Attempt to open data file 
	if (!infile) {
		std::cout << "While opening data file an error was encountered" << std::endl;
	}
	else {
		string line;
		int fileColumn, freqCount = 0;
		(*frequencyInfo).fpointCount = 0;
		bool skipline;
		string word;

		//Ali: iterate through each line
		while (getline(infile, line)) {
			istringstream stringOfLine(line);
			fileColumn = 0;

			//Ali: Iterrate through each element of line which is refered to as a column
			while (stringOfLine) {
				//load new element into a string variable called "word"
				stringOfLine >> word;
				skipline = false;

				//Ali: if comment ("%#") is detected then like is skipped
				if (word.compare("%#") == 0) {
					skipline = true;
					break;
				}

				//Ali: if it is the first column then data is stored as the frequency
				if (fileColumn == 0) {
					//Ali: translate string to int 
					freq[freqCount] = stod(word, &sz);

					//Ali: if it is the forst freq reading then initialize max and min
					if (freqCount == 0) {
						(*frequencyInfo).high = freq[freqCount];
						(*frequencyInfo).low = freq[freqCount];
					}
					else {
						//Ali: if current freq is greater than highest then update 
						if ((*frequencyInfo).high < freq[freqCount]) {
							(*frequencyInfo).high = freq[freqCount];
						}
						//Ali: if current freq is smaller than lowest then update 
						if ((*frequencyInfo).low > freq[freqCount]) {
							(*frequencyInfo).low = freq[freqCount];
						}
					}

				}
				//Ali: The second column is the real part of the response
				if (fileColumn == 1) {
					data[freqCount].x = stod(word, &sz);
				}
				//Ali: The third column is the imag part of the response
				if (fileColumn == 2) {
					data[freqCount].y = stod(word, &sz);
				}
				fileColumn++;
			}
			//Ali: If line is skipped then dont add to freq count
			if (!skipline) {
				freqCount++;
			}
		}

		//Ali: Update total freq count
		(*frequencyInfo).fpointCount = freqCount;

		//Ali: make sure the min is atleast 1e-6
		if ((*frequencyInfo).low < MINFREQ) {
			(*frequencyInfo).low = MINFREQ;
		}
	}

	if (debug) {
		printf("\n\n********************************************************\n");
		printf("extracted data\n");
		printf("********************************************************\n");
		for (int z = 0; z < (*frequencyInfo).fpointCount; z++) {
			printf("Z: %d FREQ: %f RESP: %f(%f) \n", z, freq[z], data[z].x, data[z].y);
		}
		printf("^^^^^^^^^^^^^^^^^^^\n");
		printf("HiegestFREQ: %f GHz \n LowestFREQ: %f GHz \n FreqPoints %d\n", (*frequencyInfo).high, (*frequencyInfo).low, (*frequencyInfo).fpointCount);
	}



	//###########################Initial Pole Guess########################################

	//This constant is predetermined in a paper
	int Real_part_Divisor = 100;

	int NumberOfPoles = NRealPoles + (NComplexPoles / 2);

	double *Poles_imag_part;
	hipMallocManaged(&Poles_imag_part, NumberOfPoles * sizeof(double));

	double *Poles_real_part;
	hipMallocManaged(&Poles_real_part, NumberOfPoles * sizeof(double));

	double *Real_Poles;
	hipMallocManaged(&Real_Poles, NRealPoles * sizeof(double));

	hipComplex *Complex_Poles;
	hipMallocManaged(&Complex_Poles, NComplexPoles * sizeof(hipComplex));

	hipComplex *Poles;
	hipMallocManaged(&Poles, (NRealPoles + NComplexPoles) * sizeof(hipComplex));

	int B[2] = { 1, 1 };
	int C[2] = { 1, -1 };

	double poleSpacing = ((*frequencyInfo).high - (*frequencyInfo).low) / (NumberOfPoles - 1);

	for (int z = 0; z < NumberOfPoles; z++) {
		Poles_imag_part[z] = (*frequencyInfo).low + poleSpacing*z;
		Poles_real_part[z] = -Poles_imag_part[z] / Real_part_Divisor;
	}

	//Set Real Poles
	for (int z = 0; z < NRealPoles; z++) {
		Real_Poles[z] = 2 * M_PI*Poles_real_part[z];
	}

	//Set Complex Poles
	int poleIndex = 0;
	for (int z = 0; z < NComplexPoles / 2; z++) {
		for (int i = 0; i < 2; i++) {
			Complex_Poles[poleIndex].x = 2 * M_PI*(Poles_real_part[NRealPoles + z] * B[i]);
			Complex_Poles[poleIndex].y = 2 * M_PI*(Poles_imag_part[NRealPoles + z] * C[i]);
			poleIndex++;
		}
	}

	//merge Poles into one matrix
	printf("444444\n");
	for (int z = 0; z < (NComplexPoles + NRealPoles); z++) {
		if (z < NRealPoles) {
			Poles[z].x = Real_Poles[z];
		}
		else {
			Poles[z].x = Complex_Poles[z - NRealPoles].x;
			Poles[z].y = Complex_Poles[z - NRealPoles].y;
		}
	}

	printf("444444\n");
	printf("\n\n********************************************************\n");
	printf("Initial Poles\n");
	printf("********************************************************\n");
	for (int z = 0; z < NRealPoles; z++) {
		printf("Real Pole[%d]: %f(%f) \n", z, Real_Poles[z]);
	}
	for (int z = 0; z < NComplexPoles; z++) {
		printf("Complex Pole[%d]: %f(%f) \n", z, Complex_Poles[z].x, Complex_Poles[z].y);
	}
	printf("^^^^^^^^^^^^^\n");
	for (int z = 0; z < NComplexPoles + NRealPoles; z++) {
		printf("merged Pole[%d]: %f(%f) \n", z, Poles[z].x, Poles[z].y);
	}
	printf("^^^^^^^^^^^^^\n");
	printf("NumberOfPoles: %d  \n poleSpacing: %f \n", NumberOfPoles, poleSpacing);

	//###########################Ahat set up########################################
	hipComplex *Ahat;
	//size_t pitch;
	//hipMallocPitch(&devPtr, &devPitch, Ncols * sizeof(float), Nrows);
	//hipMallocPitch(&Ahat, &pitch, Ncols * sizeof(float), Nrows));
	int NPorts = 1;
	int* Apattern;
	hipMallocManaged(&Apattern, (NRealPoles + NComplexPoles + NPorts) * sizeof(int));

	int isReal = 1;
	for (int i = 0; i < NRealPoles * 2 + NComplexPoles * 2 + NPorts; i++) {
		if (i < NRealPoles) {
			Apattern[i] = 1;
		}
		else if (i < NRealPoles + NComplexPoles) {
			Apattern[i] = (isReal) ? 2 : 3;
			isReal ^= 1;
		}
		else if (i < NRealPoles + NComplexPoles + NPorts) {
			Apattern[i] = 4;
		}
		else if ((i - NComplexPoles - NPorts - NRealPoles) < NRealPoles) {
			Apattern[i] = -1;
		}
		else if ((i - NComplexPoles - NPorts - NRealPoles) < NRealPoles + NComplexPoles) {
			Apattern[i] = (isReal) ? -2 : -3;
			isReal ^= 1;
		}
	};



	printf(" A Pattern: ");
	for (int i = 0; i < NRealPoles * 2 + NComplexPoles * 2 + NPorts; i++) {
		printf("%d ", Apattern[i]);
	};
	printf("\n");

	int NCol = NRealPoles * 2 + NComplexPoles * 2;
	int NRow = (*frequencyInfo).fpointCount;
	hipMallocManaged(&Ahat, NRow*((2 * NRealPoles + 2 * NComplexPoles + NPorts) * sizeof(double)));
	double s;
	printf("here 1\n");
	int g = 0;
	/*	for (int col = 0; col < NRealPoles; col++) {
			for (int row = 0; row < (*frequencyInfo).fpointCount; row++) {
				s = 2 * M_PI*freq[row];
				//Ahat[col*NRow + row].x = (1 / (freq[row] - Real_Poles[col]);
				Ahat[col*NRow + row].x = -Real_Poles[col]/(pow(Real_Poles[col],2) + pow(s,2));
				Ahat[col*NRow + row].y = -s / (pow(Real_Poles[col], 2) + pow(s, 2));
				g++;
			};
		}; */


	int Ahat_size = NComplexPoles * 2 + NRealPoles * 2 + NPorts;
	int CPUenable = 1;
	if(CPUenable==1){
		clock_t tStart = clock();
		/* Do your stuff here */

		

		printf("here 2\n");
		double real;
		double imag;
		double denum;
		int poleNumb = 0;
		int test = 0;
		for (int col = 0; col < Ahat_size; col++) {
			real = Poles[poleNumb].x;
			imag = Poles[poleNumb].y;

			for (int row = 0; row < (*frequencyInfo).fpointCount; row++) {
				s = 2 * M_PI*freq[row];

				//real		
				if (Apattern[col] == 1) {
					Ahat[col*NRow + row].x = -real / (pow(real, 2) + pow(s, 2));
					Ahat[col*NRow + row].y = -s / (pow(real, 2) + pow(s, 2));
				}
				else if (Apattern[col] == 2) {
					denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
					Ahat[col*NRow + row].x = -2 * (real*(pow(real, 2) + pow(s, 2) + pow(imag, 2))) / denum;
					Ahat[col*NRow + row].y = -2 * (s *(pow(real, 2) + pow(s, 2) - pow(imag, 2))) / denum;
				}
				else if (Apattern[col] == 3) {
					denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
					Ahat[(col)*NRow + row].x = (-2 * imag*(pow(real, 2) - pow(s, 2) + pow(imag, 2))) / denum;
					Ahat[(col)*NRow + row].y = (-4 * real*imag*s) / denum;
				}
				else if (Apattern[col] == 4) {
					Ahat[col*NRow + row].x = 1;
					Ahat[col*NRow + row].y = 0;
				}
				else if (Apattern[col] == -1) {
					denum = pow(real, 2) + pow(imag, 2) - 2 * imag*s + pow(s, 2);
					Ahat[col*NRow + row].x = (real*data[row].x - data[row].y*s + data[row].y*imag) / denum;
					Ahat[col*NRow + row].y = (s*data[row].x - data[row].x*imag + data[row].y*real) / denum;
				}
				else if (Apattern[col] == -2) {
					denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
					Ahat[col*NRow + row].x = 2 * (pow(real, 3)*data[row].x - pow(real, 2)*data[row].y*s + real* data[row].x*pow(s, 2) + real* data[row].x*pow(imag, 2) + s*data[row].y*pow(imag, 2) - data[row].y*pow(s, 3)) / denum;
					Ahat[col*NRow + row].y = 2 * (pow(real, 3)*data[row].y + pow(real, 2)*data[row].x*s + real* data[row].y*pow(s, 2) + real* data[row].y*pow(imag, 2) - s*data[row].x*pow(imag, 2) + data[row].x*pow(s, 3)) / denum;
				}
				else if (Apattern[col] == -3) {
					denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
					Ahat[col*NRow + row].x = 2 * imag*(pow(real, 2)*data[row].x - data[row].x* pow(s, 2) + data[row].x*pow(imag, 2) - 2 * real*data[row].y*s) / denum;
					Ahat[col*NRow + row].y = 2 * imag*(2 * data[row].x*real*s + pow(real, 2)*data[row].y - data[row].y*pow(s, 2) + data[row].y*pow(imag, 2)) / denum;
				}

				//imag			
				g++;
			};
			if (poleNumb < (NComplexPoles + NRealPoles)) {
				poleNumb++;
			}
			else {
				poleNumb = 0;
			}
		};

		clock_t tStop = clock();
		printf("CPU Time taken: %.6fs\n", (double)(tStop - tStart) / CLOCKS_PER_SEC);
	}


	clock_t start = clock();
	VectorAdd <<<Ahat_size,(*frequencyInfo).fpointCount  >>> (freq, Ahat, data, Poles, frequencyInfo, Apattern, Ahat_size, NComplexPoles, NRealPoles);
	hipDeviceSynchronize();
	clock_t stop = clock();
	printf("GPU Time taken: %.6fs\n", (double)(stop - start) / CLOCKS_PER_SEC);














	FILE * fp;

	fp = fopen("file.txt", "w+");
	for (int row = 0; row <(*frequencyInfo).fpointCount; row++) {
		for (int col = 0; col < NComplexPoles*2 + NRealPoles*2 + NPorts; col++) {
			fprintf(fp," %.4e(%.4e)", Ahat[col*NRow + row].x, Ahat[col*NRow + row].y);
		};
		fprintf(fp,"\n");
	};

	fclose(fp);

/*	Poles_imag_part = linspace(f.L, f.H, IP.Nreal + IP.Ncomplex / 2);
	Poles_real_part = -Poles_imag_part / Real_part_Divisor;

	Real_Poles = Poles_real_part(1 : IP.Nreal);

	Complex_Poles = ...
		kron(Poles_real_part(IP.Nreal + 1:end), [1, 1]) + ...
		kron(Poles_imag_part(IP.Nreal + 1:end), [-j, j]);

	initial_Poles = 2 * pi*transpose(cat(2, Real_Poles, Complex_Poles)); */

	

	hipFree(freq);
	hipFree(data);
	hipFree(frequencyInfo);
	hipFree(Poles_imag_part);
	hipFree(Poles_real_part);
	hipFree(Real_Poles);
	hipFree(Complex_Poles);

	return 0;
}