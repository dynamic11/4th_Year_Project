#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <hip/hip_complex.h>
#include "kernel.h"
#define _USE_MATH_DEFINES
#include <math.h>
#include <time.h>
#include ""

const int SIZE = 1024;
const double MINFREQ = 1e-6;
using namespace std;
bool debug;

struct dataInfo {
	double freqHigh;
	double freqLow;
	int NFreq;
	int NPorts;
};

__global__ void VectorAdd(double *freq, hipComplex *Ahat, hipComplex *data, hipComplex *Poles, dataInfo *frequencyInfo, int *Apattern, int Ahat_size, int NComplexPoles, int NRealPoles)
{
	double real=0;
	double imag=0;
	double denum=0;
	int poleNumb = 0;
	int NRow = (*frequencyInfo).NFreq;
	double s;
	int test = 0;
		int col = blockIdx.x;
		int row = threadIdx.x;
		if (col > 17) {
			poleNumb = col - 17;
		}
		else {
			poleNumb = col;
		}

		real = Poles[poleNumb].x;
		imag = Poles[poleNumb].y;

			s = 2 * M_PI*freq[row];

		//real		
		if (Apattern[col] == 1) {
			Ahat[col*NRow + row].x = -real / (pow(real, 2) + pow(s, 2));
			Ahat[col*NRow + row].y = -s / (pow(real, 2) + pow(s, 2));
		}
		else if (Apattern[col] == 2) {
			denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
			Ahat[col*NRow + row].x = -2 * (real*(pow(real, 2) + pow(s, 2) + pow(imag, 2))) / denum;
			Ahat[col*NRow + row].y = -2 * (s *(pow(real, 2) + pow(s, 2) - pow(imag, 2))) / denum;
		}
		else if (Apattern[col] == 3) {
			denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
			Ahat[(col)*NRow + row].x = (-2 * imag*(pow(real, 2) - pow(s, 2) + pow(imag, 2))) / denum;
			Ahat[(col)*NRow + row].y = (-4 * real*imag*s) / denum;
		}
		else if (Apattern[col] == 4) {
			Ahat[col*NRow + row].x = 1;
			Ahat[col*NRow + row].y = 0;
		}
		else if (Apattern[col] == -1) {
			denum = pow(real, 2) + pow(imag, 2) - 2 * imag*s + pow(s, 2);
			Ahat[col*NRow + row].x = (real*data[row].x - data[row].y*s + data[row].y*imag) / denum;
			Ahat[col*NRow + row].y = (s*data[row].x - data[row].x*imag + data[row].y*real) / denum;
		}
		else if (Apattern[col] == -2) {
			denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
			Ahat[col*NRow + row].x = 2 * (pow(real, 3)*data[row].x - pow(real, 2)*data[row].y*s + real* data[row].x*pow(s, 2) + real* data[row].x*pow(imag, 2) + s*data[row].y*pow(imag, 2) - data[row].y*pow(s, 3)) / denum;
			Ahat[col*NRow + row].y = 2 * (pow(real, 3)*data[row].y + pow(real, 2)*data[row].x*s + real* data[row].y*pow(s, 2) + real* data[row].y*pow(imag, 2) - s*data[row].x*pow(imag, 2) + data[row].x*pow(s, 3)) / denum;
		}
		else if (Apattern[col] == -3) {
			denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
			Ahat[col*NRow + row].x = 2 * imag*(pow(real, 2)*data[row].x - data[row].x* pow(s, 2) + data[row].x*pow(imag, 2) - 2 * real*data[row].y*s) / denum;
			Ahat[col*NRow + row].y = 2 * imag*(2 * data[row].x*real*s + pow(real, 2)*data[row].y - data[row].y*pow(s, 2) + data[row].y*pow(imag, 2)) / denum;
		}

}



//__global__ void AMatrixFormulationGPU(double *freq, hipComplex *Ahat, hipComplex *data, hipComplex *Poles, dataInfo *frequencyInfo, int *Apattern, int Ahat_size, int NComplexPoles, int NRealPoles) {
//	for (int blocky = 0; blocky <= NColToTake; blocky++) {
//		for (int blockx = 0; blockx < NColToTake; blockx++) {
//			if (blocky == NColToTake) {
//				for (int col = 0; col < baseMatrix_NCol; col++) {
//					for (int row = 0; row < baseMatrix_NRow; row++) {
//
//						realBase = baseMatrix[col*NRow + row].x;
//						imagBase = baseMatrix[col*NRow + row].y;
//
//						realData = data[row + NFreq*blockx].x;
//						imagData = data[row + NFreq*blockx].y;
//
//						int memPosition = blocky*(baseMatrix_NCol)*(NFreq)*(NColToTake)+col*NFreq*NColToTake + blockx*NFreq + row;
//
//						Ahat[memPosition].x = (realBase*realData - imagBase*imagData);
//						Ahat[memPosition].y = (realBase*imagData - imagBase*realData);
//					}
//				}//end for col
//			}
//			else if (blocky == blockx) {
//				for (int col = 0; col < baseMatrix_NCol; col++) {
//					for (int row = 0; row < baseMatrix_NRow; row++) {
//						int memPosition = blocky*(baseMatrix_NCol)*(NFreq)*(NColToTake)+col*NFreq*NColToTake + blockx*NFreq + row;
//						Ahat[memPosition].x = baseMatrix[col*NRow + row].x;
//						Ahat[memPosition].y = baseMatrix[col*NRow + row].y;
//					}//end for row
//				}//end for col
//			}
//			else {
//				for (int col = 0; col < baseMatrix_NCol; col++) {
//
//					for (int row = 0; row < baseMatrix_NCol; row++) {
//
//						int memPosition = blocky*(baseMatrix_NCol)*(NFreq)*(NColToTake)+col*NFreq*NColToTake + blockx*NFreq + row;
//						Ahat[memPosition].x = 0;
//						Ahat[memPosition].y = 0;
//					}//end for row
//				}//end for col
//			}//endif
//		}
//	}
//
//}


//
void readFile(string fileName, double *freq, hipComplex *data, dataInfo *dataInfo) {
	std::string::size_type sz;

	ifstream infile(fileName);

	//Ali: Attempt to open data file 
	if (!infile) {
		std::cout << "While opening data file an error was encountered" << std::endl;
	}
	else {
		string line;
		int fileColumn, freqCount = 0;
		bool skipline;
		string word;
		int currentFileDataCol = 0, currentPole = 0, startofPoleCol = 0, endofPoleCol = 0, dataType = 0;

		//Ali: iterate through each line
		while (getline(infile, line)) {
			istringstream stringOfLine(line);
			fileColumn = 0;
			int dataColCount = 0;

			//Ali: Iterrate through each element of line which is refered to as a column
			while (stringOfLine) {

				//Ali: escape after last pole
				if (fileColumn == 2 * pow((*dataInfo).NPorts, 2) + 1) {
					break;
				}

				//load new element into a string variable called "word"
				stringOfLine >> word;
				skipline = false;

				//Ali: if comment ("%#") is detected then like is skipped
				if (word.compare("%#") == 0) {
					skipline = true;
					break;
				}

				//Ali: if it is the first column then data is stored as the frequency
				if (fileColumn == 0) {
					//Ali: translate string to int 
					freq[freqCount] = stod(word, &sz);

					//Ali: if it is the forst freq reading then initialize max and min
					if (freqCount == 0) {
						(*dataInfo).freqHigh = freq[freqCount];
						(*dataInfo).freqLow = freq[freqCount];
					}
					else {
						//Ali: if current freq is greater than highest then update 
						if ((*dataInfo).freqHigh < freq[freqCount]) {
							(*dataInfo).freqHigh = freq[freqCount];
						}
						//Ali: if current freq is smaller than lowest then update 
						if ((*dataInfo).freqLow > freq[freqCount]) {
							(*dataInfo).freqLow = freq[freqCount];
						}
					}
					fileColumn++;

				}
				else {
					//Ali: stores the data coloumn number without freq coloumn
					currentFileDataCol = fileColumn - 1;
					//Ali: calculate the current port we are reading
					currentPole = (int)(currentFileDataCol / ((*dataInfo).NPorts * 2));
					//Ali: calculate the start coloumn and end coloumn to start and stop storing data for port
					startofPoleCol = currentPole * 2 * (*dataInfo).NPorts;
					endofPoleCol = startofPoleCol + (currentPole + 1) * 2 - 1;

					//Ali: check if it should store the current coloumn
					if (currentFileDataCol >= startofPoleCol && currentFileDataCol <= endofPoleCol) {
						//Ali: The second column is the real part of the response
						if (dataType == 0) {
							data[freqCount + (*dataInfo).NFreq*(dataColCount)].x = stod(word, &sz);
						}//endif
						//Ali: The third column is the imag part of the response
						if (dataType == 1) {
							data[freqCount + (*dataInfo).NFreq*(dataColCount)].y = stod(word, &sz);
							dataColCount++;
						}//endif
						dataType ^= 1;
					} //endif
					fileColumn++;
				}//endif
			}//endwhile

			//Ali: If line is skipped then dont add to freq count
			if (!skipline) {
				freqCount++;
			}
		}

		//Ali: make sure the min is atleast 1e-6
		if ((*dataInfo).freqLow < MINFREQ) {
			(*dataInfo).freqLow = MINFREQ;
		}
		printf("freqcount %d\n", (*dataInfo).NFreq);
	}
}//enfunction

int main()
{
	debug = true;
	string dataFileName = "radial_stub^S.txt";
	int NRealPoles = 1;
	int NComplexPoles = 2;
	int NPorts = 2;
	int NFreq = 1001;
	int NColOfData = 0;

	//###########################Reading File########################################

	//Ali: Find out how many col we wil have to store based on number of ports
	for (int i = 1; i <= NPorts; i++) {
		NColOfData += i;
	}

	//Ali: var to store freq points in data (upto 1024 data points)
	double *freq;
	hipMallocManaged(&freq, NFreq * sizeof(double));

	//Ali: store collected data in complex form (upto 1024 data points)
	hipComplex *data;
	hipMallocManaged(&data, NColOfData * NFreq * sizeof(hipComplex));

	//Ali: store info about the stroed data
	//		-lowest freq
	//		-highest freq
	//		-number of freq points
	dataInfo *dataInfo;
	hipMallocManaged(&dataInfo, sizeof(dataInfo));

	(*dataInfo).NFreq = NFreq;
	(*dataInfo).NPorts = NPorts;

	//Ali: extract data form file
	readFile(dataFileName, freq, data, dataInfo);


	if (debug) {

		FILE * fp;
		fp = fopen("1_extractedData.txt", "w+");

		fprintf(fp, "********************************************************\n");
		fprintf(fp, "extracted data\n");
		fprintf(fp, "********************************************************\n");
		for (int i = 0; i < NColOfData; i++) {
			fprintf(fp, "\n********************************************************\n");
			fprintf(fp, "col: %d \n", i);
			for (int z = 0; z < (*dataInfo).NFreq; z++) {
				fprintf(fp, "Z: %d FREQ: %f %f(%f) \n", z, freq[z], data[i*NFreq + z].x, data[i*NFreq + z].y);
			}
		}
		fclose(fp);
		printf("^^^^^^^^^^^^^^^^^^^\n");
		printf("HiegestFREQ: %f GHz \n LowestFREQ: %f GHz \n FreqPoints %d\n", (*dataInfo).freqHigh, (*dataInfo).freqLow, (*dataInfo).NFreq);
	}

	//###########################Initial Pole Guess########################################

	//This constant is predetermined in a paper
	int Real_part_Divisor = 100;

	int NumberOfPoles = NRealPoles + (NComplexPoles / 2);

	double *Poles_imag_part;
	hipMallocManaged(&Poles_imag_part, NumberOfPoles * sizeof(double));

	double *Poles_real_part;
	hipMallocManaged(&Poles_real_part, NumberOfPoles * sizeof(double));

	double *Real_Poles;
	hipMallocManaged(&Real_Poles, NRealPoles * sizeof(double));

	hipComplex *Complex_Poles;
	hipMallocManaged(&Complex_Poles, NComplexPoles * sizeof(hipComplex));

	hipComplex *Poles;
	hipMallocManaged(&Poles, (NRealPoles + NComplexPoles) * sizeof(hipComplex));

	int B[2] = { 1, 1 };
	int C[2] = { 1, -1 };

	double poleSpacing = ((*dataInfo).freqHigh - (*dataInfo).freqLow) / (NumberOfPoles - 1);

	for (int z = 0; z < NumberOfPoles; z++) {
		Poles_imag_part[z] = (*dataInfo).freqLow + poleSpacing*z;
		Poles_real_part[z] = -Poles_imag_part[z] / Real_part_Divisor;
	}

	//Set Real Poles
	for (int z = 0; z < NRealPoles; z++) {
		Real_Poles[z] = 2 * M_PI*Poles_real_part[z];
	}

	//Set Complex Poles
	int poleIndex = 0;
	for (int z = 0; z < NComplexPoles / 2; z++) {
		for (int i = 0; i < 2; i++) {
			Complex_Poles[poleIndex].x = 2 * M_PI*(Poles_real_part[NRealPoles + z] * B[i]);
			Complex_Poles[poleIndex].y = 2 * M_PI*(Poles_imag_part[NRealPoles + z] * C[i]);
			poleIndex++;
		}
	}

	//merge Poles into one matrix
	for (int z = 0; z < (NComplexPoles + NRealPoles); z++) {
		if (z < NRealPoles) {
			Poles[z].x = Real_Poles[z];
		}
		else {
			Poles[z].x = Complex_Poles[z - NRealPoles].x;
			Poles[z].y = Complex_Poles[z - NRealPoles].y;
		}
	}

	printf("\n\n********************************************************\n");
	printf("Initial Poles\n");
	printf("********************************************************\n");
	for (int z = 0; z < NRealPoles; z++) {
		printf("Real Pole[%d]: %f(%f) \n", z, Real_Poles[z]);
	}
	for (int z = 0; z < NComplexPoles; z++) {
		printf("Complex Pole[%d]: %f(%f) \n", z, Complex_Poles[z].x, Complex_Poles[z].y);
	}
	printf("^^^^^^^^^^^^^\n");
	for (int z = 0; z < NComplexPoles + NRealPoles; z++) {
		printf("merged Pole[%d]: %f(%f) \n", z, Poles[z].x, Poles[z].y);
	}
	printf("^^^^^^^^^^^^^\n");
	printf("NumberOfPoles: %d  \n poleSpacing: %f \n", NumberOfPoles, poleSpacing);





	//########################### Base Matrix Setup ########################################

	// NCol is equal to the number of real poles + number of imaginary poles + 1 (d col)
	int baseMatrix_NCol = NComplexPoles + NRealPoles + 1;
	int baseMatrix_NRow = (*dataInfo).NFreq;

	int NRow = (*dataInfo).NFreq;

	hipComplex *baseMatrix;
	hipMallocManaged(&baseMatrix, baseMatrix_NRow * baseMatrix_NCol * sizeof(double));

	int CPUenable = 1;
	if (CPUenable == 1 || CPUenable == 2) {
		clock_t tStart = clock();
		/* Do your stuff here */

		double real = 0, imag = 0, denum = 0, s=0;
		int poleNumb = 0, isReal = 1;
	
		//for loop to generate base matrix
		for (int col = 0; col < baseMatrix_NCol; col++) {

			if (poleNumb < NRealPoles + NComplexPoles) {
				real = Poles[poleNumb].x;
				imag = Poles[poleNumb].y;
			}

			for (int row = 0; row < baseMatrix_NRow; row++) {
				s = 2 * M_PI*freq[row];
				//real pole		
				if (col < NRealPoles) {
					baseMatrix[col*NRow + row].x = -real / (pow(real, 2) + pow(s, 2));
					baseMatrix[col*NRow + row].y = -s / (pow(real, 2) + pow(s, 2));

				}
				//imag pole	real part
				else if (col < NRealPoles + NComplexPoles && isReal) {
					denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
					baseMatrix[col*NRow + row].x = -2 * (real*(pow(real, 2) + pow(s, 2) + pow(imag, 2))) / denum;
					baseMatrix[col*NRow + row].y = -2 * (s *(pow(real, 2) + pow(s, 2) - pow(imag, 2))) / denum;
				}
				//imag pole	imag part
				else if (col < NRealPoles + NComplexPoles && !isReal) {
					denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
					baseMatrix[col*NRow + row].x = (-2 * imag*(pow(real, 2) - pow(s, 2) + pow(imag, 2))) / denum;
					baseMatrix[col*NRow + row].y = (-4 * real*imag*s) / denum;
				}
				//d col
				else if (col == NRealPoles + NComplexPoles) {
					baseMatrix[col*NRow + row].x = 1;
					baseMatrix[col*NRow + row].y = 0;
				}//endif
			};//end row for loop
			
			poleNumb++;

			if(col>= NRealPoles && col < NRealPoles + NComplexPoles)
				isReal ^= 1;
		}; //end col for loop

		//Write base b matrix to file
		FILE * fp;
		fp = fopen("2_Basematrix.txt", "w+");
		for (int row = 0; row < baseMatrix_NRow; row++) {
			for (int col = 0; col < baseMatrix_NCol; col++) {
				fprintf(fp, " %.4e(%.4e)", baseMatrix[col*NRow + row].x, baseMatrix[col*NRow + row].y);
			};
			fprintf(fp, "\n");
		};
		fclose(fp);


		double realBase, imagBase, realData, imagData;
		int dim = NColOfData;


		//###########################Ahat set up########################################
		//Ali: generate the AhatMatrix
		hipComplex *Ahat;
		hipMallocManaged(&Ahat, (dim)*NRow*(baseMatrix_NCol)*(dim+1) * sizeof(double));
		poleNumb = 0;


		for (int blocky = 0; blocky <= NColOfData; blocky++) {
			for (int blockx = 0; blockx < NColOfData; blockx++) {
				if (blocky == NColOfData) {
					for (int col = 0; col < baseMatrix_NCol; col++) {
						for (int row = 0; row < baseMatrix_NRow; row++) {

							realBase = baseMatrix[col*NRow + row].x;
							imagBase = baseMatrix[col*NRow + row].y;

							realData = data[row + NFreq*blockx].x;
							imagData = data[row + NFreq*blockx].y;

							int memPosition = blocky*(baseMatrix_NCol)*(NFreq)*(NColOfData)+col*NFreq*NColOfData + blockx*NFreq + row;

							Ahat[memPosition].x = (realBase*realData - imagBase*imagData);
							Ahat[memPosition].y = (realBase*imagData - imagBase*realData);
						}
					}//end for col
				}
				else if (blocky == blockx) {
					for (int col = 0; col < baseMatrix_NCol; col++) {
						for (int row = 0; row < baseMatrix_NRow; row++) {
							int memPosition = blocky*(baseMatrix_NCol)*(NFreq)*(NColOfData)+col*NFreq*NColOfData + blockx*NFreq + row;
							Ahat[memPosition].x = baseMatrix[col*NRow + row].x;
							Ahat[memPosition].y = baseMatrix[col*NRow + row].y;
						}//end for row
					}//end for col
				}
				else {
					for (int col = 0; col < baseMatrix_NCol; col++) {

						for (int row = 0; row < baseMatrix_NCol; row++) {
							
							int memPosition = blocky*(baseMatrix_NCol)*(NFreq)*(NColOfData)+col*NFreq*NColOfData + blockx*NFreq + row;
							Ahat[memPosition].x = 0;
							Ahat[memPosition].y = 0;
						}//end for row
					}//end for col
				}//endif
			}
		}


		
		clock_t tStop = clock();
		printf("CPU Time taken: %.6fs\n", (double)(tStop - tStart) / CLOCKS_PER_SEC);



	//clock_t start = clock();
	//VectorAdd <<<Ahat_size,(*dataInfo).NFreq  >>> (freq, Ahat, data, Poles, dataInfo, Apattern, Ahat_size, NComplexPoles, NRealPoles);
	//hipDeviceSynchronize();
	//clock_t stop = clock();
	//printf("GPU Time taken: %.6fs\n", (double)(stop - start) / CLOCKS_PER_SEC);


	//FILE * fp;

	fp = fopen("2_Amatrix.txt", "w+");
	for (int row = 0; row < NColOfData*NFreq; row++) {
		for (int col = 0; col < baseMatrix_NCol*(NColOfData+1); col++) {
			fprintf(fp, " %.4e(%.4e)", Ahat[col*NRow*NColOfData + row].x, Ahat[col*NRow*NColOfData + row].y);
		}
		fprintf(fp, "\n");
	}



	//for (int blocky = 0; blocky <= NColToTake; blocky++) {
	//	for (int blockx = 0; blockx < NColToTake; blockx++) {
	//		for (int row = 0; row < baseMatrix_NRow; row++) {
	//			for (int col = 0; col < baseMatrix_NCol; col++) {
	//				int memPosition = blocky*(baseMatrix_NCol)*(NFreq)*(NColToTake)+col*NFreq*NColToTake + blockx*NFreq + row;
	//				fprintf(fp, " %.4e(%.4e)", Ahat[memPosition].x, Ahat[memPosition].y);
	//			};
	//			fprintf(fp, "\n");
	//		};
	//	}
	//}
	fclose(fp);
}

/*	Poles_imag_part = linspace(f.L, f.H, IP.Nreal + IP.Ncomplex / 2);
	Poles_real_part = -Poles_imag_part / Real_part_Divisor;

	Real_Poles = Poles_real_part(1 : IP.Nreal);

	Complex_Poles = ...
		kron(Poles_real_part(IP.Nreal + 1:end), [1, 1]) + ...
		kron(Poles_imag_part(IP.Nreal + 1:end), [-j, j]);

	initial_Poles = 2 * pi*transpose(cat(2, Real_Poles, Complex_Poles)); */

	

	hipFree(freq);
	hipFree(data);
	hipFree(dataInfo);
	hipFree(Poles_imag_part);
	hipFree(Poles_real_part);
	hipFree(Real_Poles);
	hipFree(Complex_Poles);

	return 0;
}