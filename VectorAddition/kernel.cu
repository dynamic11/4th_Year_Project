#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <hip/hip_complex.h>
#include "kernel.h"
#define _USE_MATH_DEFINES
#include <math.h>

const int SIZE = 1024;
const double MINFREQ = 1e-6;
using namespace std;
bool debug;

//__global__ void VectorAdd(int *a, int *b, int *c, int n)
//{
//	int i = threadIdx.x;
//	if(i < n)
//		c[i] = a[i] + b[i];
//}
//
//void readFile(char *fileName, int *data)
//{
//	int i = threadIdx.x;
//	if (i < n)
//		c[i] = a[i] + b[i];
//}

int main()
{
	debug = true;
	string dataFileName = "radial_stub^S.txt";
	int NRealPoles = 1;
	int NComplexPoles = 16;

	//###########################Reading File########################################

	

	//Ali: var to store freq points in data (upto 1024 data points)
	double *freq;
	hipMallocManaged(&freq, SIZE * sizeof(double));

	//Ali: store collected data in complex form (upto 1024 data points)
	hipComplex *data;
	hipMallocManaged(&data, SIZE * sizeof(hipComplex));

	//Ali: store info about the stroed data
	//		-lowest freq
	//		-highest freq
	//		-number of freq points
	frequency *frequencyInfo;
	hipMallocManaged(&frequencyInfo, sizeof(frequency));

	std::string::size_type sz;
	

	ifstream infile(dataFileName);
	
	//Ali: Attempt to open data file 
	if (!infile) {
		std::cout << "While opening data file an error was encountered" << std::endl;
	} else {
		string line;
		int fileColumn, freqCount = 0;
		(*frequencyInfo).fpointCount = 0;
		bool skipline;
		string word;

		//Ali: iterate through each line
		while (getline(infile, line)) {
			istringstream stringOfLine(line);
			fileColumn = 0;

			//Ali: Iterrate through each element of line which is refered to as a column
			while (stringOfLine) {
				//load new element into a string variable called "word"
				stringOfLine >> word;
				skipline = false;

				//Ali: if comment ("%#") is detected then like is skipped
				if (word.compare("%#") == 0) {
					skipline = true;
					break;
				}

				//Ali: if it is the first column then data is stored as the frequency
				if (fileColumn == 0) {
					//Ali: translate string to int 
					freq[freqCount] = stod(word, &sz);

					//Ali: if it is the forst freq reading then initialize max and min
					if (freqCount == 0) {
						(*frequencyInfo).high = freq[freqCount];
						(*frequencyInfo).low = freq[freqCount];
					}else {
						//Ali: if current freq is greater than highest then update 
						if ((*frequencyInfo).high < freq[freqCount]) {
							(*frequencyInfo).high = freq[freqCount];
						}
						//Ali: if current freq is smaller than lowest then update 
						if ((*frequencyInfo).low > freq[freqCount]) {
							(*frequencyInfo).low = freq[freqCount];
						}	
					}

				}
				//Ali: The second column is the real part of the response
				if (fileColumn == 1) {
					data[freqCount].x = stod(word, &sz);
				}
				//Ali: The third column is the imag part of the response
				if (fileColumn == 2) {
					data[freqCount].y = stod(word, &sz);
				}
				fileColumn++;
			}
			//Ali: If line is skipped then dont add to freq count
			if (!skipline) {
				freqCount++;
			}		
		}

		//Ali: Update total freq count
		(*frequencyInfo).fpointCount= freqCount;

		//Ali: make sure the min is atleast 1e-6
		if ((*frequencyInfo).low < MINFREQ) {
			(*frequencyInfo).low = MINFREQ;
		}
	}

	if (debug) {
		printf("\n\n********************************************************\n");
		printf("extracted data\n");
		printf("********************************************************\n");
		for (int z = 0; z < (*frequencyInfo).fpointCount; z++) {
			printf("Z: %d FREQ: %f RESP: %f(%f) \n", z, freq[z], data[z].x, data[z].y);
		}
		printf("^^^^^^^^^^^^^^^^^^^\n");
		printf("HiegestFREQ: %f GHz \n LowestFREQ: %f GHz \n FreqPoints %d\n", (*frequencyInfo).high, (*frequencyInfo).low, (*frequencyInfo).fpointCount);
	}



	//###########################Initial Pole Guess########################################

	//This constant is predetermined in a paper
	int Real_part_Divisor = 100;

	int NumberOfPoles = NRealPoles + (NComplexPoles / 2);

	double *Poles_imag_part;
	hipMallocManaged(&Poles_imag_part, NumberOfPoles * sizeof(double));

	double *Poles_real_part;
	hipMallocManaged(&Poles_real_part, NumberOfPoles * sizeof(double));

	double *Real_Poles;
	hipMallocManaged(&Real_Poles, NRealPoles * sizeof(double));

	hipComplex *Complex_Poles;
	hipMallocManaged(&Complex_Poles, NComplexPoles * sizeof(hipComplex));

	hipComplex *Poles;
	hipMallocManaged(&Poles, (NRealPoles+ NComplexPoles) * sizeof(hipComplex));

	int B[2] = { 1, 1 };
	int C[2] = { 1, -1 };

	double poleSpacing = ((*frequencyInfo).high - (*frequencyInfo).low) / (NumberOfPoles-1);

	for (int z = 0; z < NumberOfPoles; z++) {
		Poles_imag_part[z] = (*frequencyInfo).low + poleSpacing*z;
		Poles_real_part[z] = -Poles_imag_part[z] / Real_part_Divisor;
	}

	//Set Real Poles
	for (int z = 0; z < NRealPoles; z++) {
		Real_Poles[z] = 2*M_PI*Poles_real_part[z];
	}
	
	//Set Complex Poles
	int poleIndex = 0;
	for (int z = 0; z < NComplexPoles/2; z++) {
		for (int i = 0; i< 2; i++) {
			Complex_Poles[poleIndex].x = 2*M_PI*(Poles_real_part[NRealPoles + z] * B[i]);
			Complex_Poles[poleIndex].y = 2*M_PI*(Poles_imag_part[NRealPoles + z] * C[i]);
			poleIndex++;
		}
	}

	//merge Poles into one matrix
	printf("444444\n");
	for (int z = 0; z < (NComplexPoles+ NRealPoles); z++) {
		if (z < NRealPoles) {
			Poles[z].x = Real_Poles[z];
		}else {
			Poles[z].x = Complex_Poles[z - NRealPoles].x;
			Poles[z].y = Complex_Poles[z - NRealPoles].y;
		}
	}

	printf("444444\n");
	printf("\n\n********************************************************\n");
	printf("Initial Poles\n");
	printf("********************************************************\n");
	for (int z = 0; z < NRealPoles; z++) {
		printf("Real Pole[%d]: %f(%f) \n", z, Real_Poles[z]);
	}
	for (int z = 0; z <  NComplexPoles; z++) {
		printf("Complex Pole[%d]: %f(%f) \n", z, Complex_Poles[z].x, Complex_Poles[z].y);
	}
	printf("^^^^^^^^^^^^^\n");
	for (int z = 0; z < NComplexPoles + NRealPoles; z++) {
		printf("merged Pole[%d]: %f(%f) \n", z, Poles[z].x, Poles[z].y);
	}
	printf("^^^^^^^^^^^^^\n");
	printf("NumberOfPoles: %d  \n poleSpacing: %f \n", NumberOfPoles, poleSpacing);

	//###########################Ahat set up########################################
	hipComplex *Ahat;
	//size_t pitch;
	//hipMallocPitch(&devPtr, &devPitch, Ncols * sizeof(float), Nrows);
	//hipMallocPitch(&Ahat, &pitch, Ncols * sizeof(float), Nrows));
	int NPorts = 1;
	int* Apattern;
	hipMallocManaged(&Apattern, (NRealPoles + NComplexPoles+ NPorts) * sizeof(int));

	int isReal = 1;
	for (int i = 0; i < NRealPoles*2 + NComplexPoles * 2 + NPorts; i++) {
		if (i < NRealPoles) {
			Apattern[i]=1;	
		}else if (i < NRealPoles+NComplexPoles) {
			Apattern[i] = (isReal) ? 2 : 3;
			isReal ^= 1;	
		}else if (i < NRealPoles + NComplexPoles+NPorts) {
			Apattern[i] = 4;
		}else if ((i- NComplexPoles - NPorts- NRealPoles) < NRealPoles) {
			Apattern[i] = -1;
		}
		else if ((i - NComplexPoles - NPorts - NRealPoles)  < NRealPoles + NComplexPoles) {
			Apattern[i] = (isReal) ? -2 : -3;
			isReal ^= 1;
		}
	};



	printf(" A Pattern: ");
	for (int i = 0; i < NRealPoles*2 + NComplexPoles * 2 + NPorts; i++) {
			printf("%d ", Apattern[i]);
	};
	printf("\n");

	int NCol = NRealPoles * 2 + NComplexPoles * 2;
	int NRow = (*frequencyInfo).fpointCount;
	hipMallocManaged(&Ahat, NRow*((2*NRealPoles + 2*NComplexPoles+NPorts) * sizeof(double)));
	double s;
	printf("here 1\n");
	int g = 0;
/*	for (int col = 0; col < NRealPoles; col++) {
		for (int row = 0; row < (*frequencyInfo).fpointCount; row++) {
			s = 2 * M_PI*freq[row];
			//Ahat[col*NRow + row].x = (1 / (freq[row] - Real_Poles[col]);
			Ahat[col*NRow + row].x = -Real_Poles[col]/(pow(Real_Poles[col],2) + pow(s,2));
			Ahat[col*NRow + row].y = -s / (pow(Real_Poles[col], 2) + pow(s, 2));
			g++;
		};
	}; */

	printf("here 2\n");
	double real;
	double imag;
	double denum;
	int poleNumb = 0;
	int test = 0;
	for (int col = 0; col < NComplexPoles*2+NRealPoles*2+NPorts; col++) {
		real = Poles[poleNumb].x;
		imag = Poles[poleNumb].y;

		printf("col: %d poleNumb: %d \n", col, poleNumb);
		for (int row = 0; row < (*frequencyInfo).fpointCount; row++) {
			s = 2 * M_PI*freq[row];
			
			//real		
			if(Apattern[col] == 1){
				Ahat[col*NRow + row].x = -real / (pow(real, 2) + pow(s, 2));
				Ahat[col*NRow + row].y = -s / (pow(real, 2) + pow(s, 2));
			}else if (Apattern[col] == 2) {
				denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
				Ahat[col*NRow + row].x = -2 * (real*(pow(real, 2) + pow(s, 2) + pow(imag, 2))) / denum;
				Ahat[col*NRow + row].y = -2 * (s *(pow(real, 2) + pow(s, 2) - pow(imag, 2))) / denum;
			}else if (Apattern[col] == 3) {
				denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
				Ahat[(col)*NRow + row].x = (-2 * imag*(pow(real, 2) - pow(s, 2) + pow(imag, 2))) / denum;
				Ahat[(col)*NRow + row].y = (-4 * real*imag*s) / denum;
			}else if (Apattern[col] == 4) {
				Ahat[col*NRow + row].x = 1;
				Ahat[col*NRow + row].y = 0;
			}
			else if (Apattern[col] == -1) {
				denum = pow(real, 2) + pow(imag, 2) - 2 * imag*s + pow(s, 2);
				Ahat[col*NRow + row].x = (real*data[row].x- data[row].y*s+ data[row].y*imag) / denum;
				Ahat[col*NRow + row].y = (s*data[row].x - data[row].x*imag + data[row].y*real) / denum;
			}
			else if (Apattern[col] == -2) {
				denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
				Ahat[col*NRow + row].x = 2 * (pow(real, 3)*data[row].x - pow(real, 2)*data[row].y*s + real* data[row].x*pow(s, 2) + real* data[row].x*pow(imag, 2) + s*data[row].y*pow(imag, 2) - data[row].y*pow(s, 3)) / denum;
				Ahat[col*NRow + row].y = 2 * (pow(real, 3)*data[row].y + pow(real, 2)*data[row].x*s + real* data[row].y*pow(s, 2) + real* data[row].y*pow(imag, 2) - s*data[row].x*pow(imag, 2) + data[row].x*pow(s, 3)) / denum;
			}
			else if (Apattern[col] == -3) {
				denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
				Ahat[col*NRow + row].x = 2*imag*(pow(real, 2)*data[row].x - data[row].x* pow(s, 2) + data[row].x*pow(imag, 2) -2*real*data[row].y*s) / denum;
				Ahat[col*NRow + row].y = 2 *imag*(2*data[row].x*real*s + pow(real, 2)*data[row].y - data[row].y*pow(s, 2) + data[row].y*pow(imag,2))  / denum;
			}

			//imag			
			g++;		
		};
		if (poleNumb < (NComplexPoles + NRealPoles)) {
			poleNumb++;
		}
		else {
			poleNumb = 0;
		}
	};

	FILE * fp;

	fp = fopen("file.txt", "w+");
	for (int row = 0; row <(*frequencyInfo).fpointCount; row++) {
		for (int col = 0; col < NComplexPoles*2 + NRealPoles*2 + NPorts; col++) {
			fprintf(fp," %.4e(%.4e)", Ahat[col*NRow + row].x, Ahat[col*NRow + row].y);
		};
		fprintf(fp,"\n");
	};

	fclose(fp);

	return(0);
	


/*	Poles_imag_part = linspace(f.L, f.H, IP.Nreal + IP.Ncomplex / 2);
	Poles_real_part = -Poles_imag_part / Real_part_Divisor;

	Real_Poles = Poles_real_part(1 : IP.Nreal);

	Complex_Poles = ...
		kron(Poles_real_part(IP.Nreal + 1:end), [1, 1]) + ...
		kron(Poles_imag_part(IP.Nreal + 1:end), [-j, j]);

	initial_Poles = 2 * pi*transpose(cat(2, Real_Poles, Complex_Poles)); */

	

	hipFree(freq);
	hipFree(data);
	hipFree(frequencyInfo);
	hipFree(Poles_imag_part);
	hipFree(Poles_real_part);
	hipFree(Real_Poles);
	hipFree(Complex_Poles);

	return 0;
}