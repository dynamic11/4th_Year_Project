
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <hip/hip_complex.h>
#define _USE_MATH_DEFINES
#include <math.h>

const int SIZE = 1024;
const int MINFREQ = (1 * pow(10, -6));
using namespace std;

//__global__ void VectorAdd(int *a, int *b, int *c, int n)
//{
//	int i = threadIdx.x;
//	if(i < n)
//		c[i] = a[i] + b[i];
//}
//
//void readFile(char *fileName, int *data)
//{
//	int i = threadIdx.x;
//	if (i < n)
//		c[i] = a[i] + b[i];
//}

int main()
{

		double *freq;
		hipMallocManaged(&freq, SIZE * sizeof(double));

		struct response {
			double real;
			double imag;
		};


		response *data;
		hipMallocManaged(&data, SIZE * sizeof(response));

		struct frequency {
			double high;
			double low;
			int fpointCount;
		};

		frequency *frequencyInfo;
		hipMallocManaged(&frequencyInfo, sizeof(frequency));

		std::string::size_type sz;
		int skipline = 0;

	ifstream infile("radial_stub^S.txt");
	
	if (!infile)
	{
		std::cout << "While opening data file an error was encountered" << std::endl;
	} else {
		string line;
		int h = 0;
		(*frequencyInfo).fpointCount = 0;
		int w = 0;
		while (getline(infile, line)) {
			istringstream iss(line);
			int i = 1;			
			while (iss) {
				string word;
				iss >> word;
				skipline = 0;
				if (word.compare("%#") == 0) {
					skipline = 1;
					break;
				}
				if (i == 1) {
					freq[h] = stod(word, &sz);
					(*frequencyInfo).fpointCount++;

					if (w == 0) {
						(*frequencyInfo).high = freq[h];
						(*frequencyInfo).low = freq[h];
						w++;
					}
					else {
						if ((*frequencyInfo).high < freq[h]) {
							(*frequencyInfo).high = freq[h];
						}
							

						if ((*frequencyInfo).low > freq[h]) {
							(*frequencyInfo).low = freq[h];
						}	
					}

				}
				if (i == 2) {
					data[h].real = stod(word, &sz);
				}
				if (i == 3) {
					data[h].imag = stod(word, &sz);
				}
				i++;
			}
			if (skipline == 0) {
				h++;
			}		
		}

		if ((*frequencyInfo).low < MINFREQ) {
			(*frequencyInfo).low = MINFREQ;
		}
	}

	for (int z = 0; z < (*frequencyInfo).fpointCount; z++) {
		printf("Z: %d FREQ: %f RESP: %f(%f) \n", z, freq[z], data[z].real, data[z].imag);
	}
	printf("********************************************************\n");
	printf("HiegestFREQ: %f GHz \n LowestFREQ: %f GHz \n FreqPoints %d\n", (*frequencyInfo).high, (*frequencyInfo).low, (*frequencyInfo).fpointCount);

	//##################################################################################

	int Real_part_Divisor = 100;
	int NRealPoles = 1;
	int NComplexPoles = 16;

	int NumberOfPoles = NRealPoles + (NComplexPoles / 2);

	double *Poles_imag_part;
	hipMallocManaged(&Poles_imag_part, NumberOfPoles * sizeof(double));

	double *Poles_real_part;
	hipMallocManaged(&Poles_real_part, NumberOfPoles * sizeof(double));

	double poleSpacing=((*frequencyInfo).high - (*frequencyInfo).low) / (NumberOfPoles-1);


	for (int z = 0; z < NumberOfPoles; z++) {
		Poles_imag_part[z] = (*frequencyInfo).low + poleSpacing*z;
		Poles_real_part[z] = Poles_imag_part[z] / Real_part_Divisor;
	}

	double *Real_Poles;
	hipMallocManaged(&Real_Poles, NRealPoles * sizeof(double));

	for (int z = 0; z < NRealPoles; z++) {
		Real_Poles[z] = Poles_real_part[z];
	}

	hipComplex *Complex_Poles;
	hipMallocManaged(&Complex_Poles, NComplexPoles * sizeof(hipComplex));

	int B[2] = { 1, 1 };
	int C[2] = { -1, 1 };

	int o = 0;
	for (int z = 0; z < NComplexPoles/2; z++) {
		for (int p = 0; p< 2; p++) {
			Complex_Poles[o].x = 2*M_PI*(Poles_imag_part[NRealPoles + z] * B[p]);
			Complex_Poles[o].y = 2*M_PI*(Poles_imag_part[NRealPoles + z] * C[p]);
			o++;
		}
	}


	for (int z = 0; z < NumberOfPoles; z++) {
		printf("Pole[%d]: %f (%f)  \n", z, Poles_imag_part[z], Poles_real_part[z]);
	}
	printf("********************************************************\n");
	printf("NumberOfPoles: %d GHz \n poleSpacing: %f \n", NumberOfPoles, poleSpacing);

	for (int z = 0; z <  NComplexPoles; z++) {
		printf("Complex Pole[%d]: %f(%f) \n", z, Complex_Poles[z].x, Complex_Poles[z].y);
	}
	printf("********************************************************\n");
	printf("NumberOfPoles: %d GHz \n poleSpacing: %f \n", NumberOfPoles, poleSpacing);


/*	Poles_imag_part = linspace(f.L, f.H, IP.Nreal + IP.Ncomplex / 2);
	Poles_real_part = -Poles_imag_part / Real_part_Divisor;

	Real_Poles = Poles_real_part(1 : IP.Nreal);

	Complex_Poles = ...
		kron(Poles_real_part(IP.Nreal + 1:end), [1, 1]) + ...
		kron(Poles_imag_part(IP.Nreal + 1:end), [-j, j]);

	initial_Poles = 2 * pi*transpose(cat(2, Real_Poles, Complex_Poles)); */

	

	hipFree(freq);
	hipFree(data);
	hipFree(frequencyInfo);

	return 0;
}