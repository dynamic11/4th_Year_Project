#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <hip/hip_complex.h>
#include "kernel.h"
#define _USE_MATH_DEFINES
#include <math.h>

const int SIZE = 1024;
const int MINFREQ = (1 * pow(10, -6));
using namespace std;

//__global__ void VectorAdd(int *a, int *b, int *c, int n)
//{
//	int i = threadIdx.x;
//	if(i < n)
//		c[i] = a[i] + b[i];
//}
//
//void readFile(char *fileName, int *data)
//{
//	int i = threadIdx.x;
//	if (i < n)
//		c[i] = a[i] + b[i];
//}

int main()
{
	string dataFileName = "radial_stub^S.txt";

	//Ali: var to store freq points in data (upto 1024 data points)
	double *freq;
	hipMallocManaged(&freq, SIZE * sizeof(double));

	//Ali: store collected data in complex form (upto 1024 data points)
	hipComplex *data;
	hipMallocManaged(&data, SIZE * sizeof(hipComplex));

	//Ali: store info about the stroed data
	//		-lowest freq
	//		-highest freq
	//		-number of freq points
	frequency *frequencyInfo;
	hipMallocManaged(&frequencyInfo, sizeof(frequency));

	std::string::size_type sz;
	

	ifstream infile(dataFileName);
	
	//Ali: Attempt to open data file 
	if (!infile) {
		std::cout << "While opening data file an error was encountered" << std::endl;
	} else {
		string line;
		int fileColumn, freqCount = 0;
		(*frequencyInfo).fpointCount = 0;
		bool skipline;
		string word;

		//Ali: iterate through each line
		while (getline(infile, line)) {
			istringstream stringOfLine(line);
			fileColumn = 0;

			//Ali: Iterrate through each element of line which is refered to as a column
			while (stringOfLine) {
				//load new element into a string variable called "word"
				stringOfLine >> word;
				skipline = false;

				//Ali: if comment ("%#") is detected then like is skipped
				if (word.compare("%#") == 0) {
					skipline = true;
					break;
				}

				//Ali: if it is the first column then data is stored as the frequency
				if (fileColumn == 0) {
					//Ali: translate string to int 
					freq[freqCount] = stod(word, &sz);

					//Ali: if it is the forst freq reading then initialize max and min
					if (freqCount == 0) {
						(*frequencyInfo).high = freq[freqCount];
						(*frequencyInfo).low = freq[freqCount];
					}else {
						//Ali: if current freq is greater than highest then update 
						if ((*frequencyInfo).high < freq[freqCount]) {
							(*frequencyInfo).high = freq[freqCount];
						}
						//Ali: if current freq is smaller than lowest then update 
						if ((*frequencyInfo).low > freq[freqCount]) {
							(*frequencyInfo).low = freq[freqCount];
						}	
					}

				}
				//Ali: The second column is the real part of the response
				if (fileColumn == 1) {
					data[freqCount].x = stod(word, &sz);
				}
				//Ali: The third column is the imag part of the response
				if (fileColumn == 2) {
					data[freqCount].y = stod(word, &sz);
				}
				fileColumn++;
			}
			//Ali: If line is skipped then dont add to freq count
			if (!skipline) {
				freqCount++;
			}		
		}

		//Ali: Update total freq count
		(*frequencyInfo).fpointCount= freqCount;

		//Ali: make sure the min is atleast 1e-6
		if ((*frequencyInfo).low < MINFREQ) {
			(*frequencyInfo).low = MINFREQ;
		}
	}

	for (int z = 0; z < (*frequencyInfo).fpointCount; z++) {
		printf("Z: %d FREQ: %f RESP: %f(%f) \n", z, freq[z], data[z].x, data[z].y);
	}
	printf("********************************************************\n");
	printf("HiegestFREQ: %f GHz \n LowestFREQ: %f GHz \n FreqPoints %d\n", (*frequencyInfo).high, (*frequencyInfo).low, (*frequencyInfo).fpointCount);

	//##################################################################################

	int Real_part_Divisor = 100;
	int NRealPoles = 1;
	int NComplexPoles = 16;

	int NumberOfPoles = NRealPoles + (NComplexPoles / 2);

	double *Poles_imag_part;
	hipMallocManaged(&Poles_imag_part, NumberOfPoles * sizeof(double));

	double *Poles_real_part;
	hipMallocManaged(&Poles_real_part, NumberOfPoles * sizeof(double));

	double poleSpacing=((*frequencyInfo).high - (*frequencyInfo).low) / (NumberOfPoles-1);


	for (int z = 0; z < NumberOfPoles; z++) {
		Poles_imag_part[z] = (*frequencyInfo).low + poleSpacing*z;
		Poles_real_part[z] = Poles_imag_part[z] / Real_part_Divisor;
	}

	double *Real_Poles;
	hipMallocManaged(&Real_Poles, NRealPoles * sizeof(double));

	for (int z = 0; z < NRealPoles; z++) {
		Real_Poles[z] = Poles_real_part[z];
	}

	hipComplex *Complex_Poles;
	hipMallocManaged(&Complex_Poles, NComplexPoles * sizeof(hipComplex));

	int B[2] = { 1, 1 };
	int C[2] = { -1, 1 };

	int o = 0;
	for (int z = 0; z < NComplexPoles/2; z++) {
		for (int p = 0; p< 2; p++) {
			Complex_Poles[o].x = 2*M_PI*(Poles_imag_part[NRealPoles + z] * B[p]);
			Complex_Poles[o].y = 2*M_PI*(Poles_imag_part[NRealPoles + z] * C[p]);
			o++;
		}
	}


	for (int z = 0; z < NumberOfPoles; z++) {
		printf("Pole[%d]: %f (%f)  \n", z, Poles_imag_part[z], Poles_real_part[z]);
	}
	printf("********************************************************\n");
	printf("NumberOfPoles: %d GHz \n poleSpacing: %f \n", NumberOfPoles, poleSpacing);

	for (int z = 0; z <  NComplexPoles; z++) {
		printf("Complex Pole[%d]: %f(%f) \n", z, Complex_Poles[z].x, Complex_Poles[z].y);
	}
	printf("********************************************************\n");
	printf("NumberOfPoles: %d GHz \n poleSpacing: %f \n", NumberOfPoles, poleSpacing);


/*	Poles_imag_part = linspace(f.L, f.H, IP.Nreal + IP.Ncomplex / 2);
	Poles_real_part = -Poles_imag_part / Real_part_Divisor;

	Real_Poles = Poles_real_part(1 : IP.Nreal);

	Complex_Poles = ...
		kron(Poles_real_part(IP.Nreal + 1:end), [1, 1]) + ...
		kron(Poles_imag_part(IP.Nreal + 1:end), [-j, j]);

	initial_Poles = 2 * pi*transpose(cat(2, Real_Poles, Complex_Poles)); */

	

	hipFree(freq);
	hipFree(data);
	hipFree(frequencyInfo);

	return 0;
}