#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <hip/hip_complex.h>
#include "kernel.h"
#define _USE_MATH_DEFINES
#include <math.h>

const int SIZE = 1024;
const double MINFREQ = 1e-6;
using namespace std;
bool debug;

//__global__ void VectorAdd(int *a, int *b, int *c, int n)
//{
//	int i = threadIdx.x;
//	if(i < n)
//		c[i] = a[i] + b[i];
//}
//
//void readFile(char *fileName, int *data)
//{
//	int i = threadIdx.x;
//	if (i < n)
//		c[i] = a[i] + b[i];
//}

int main()
{
	debug = true;
	string dataFileName = "radial_stub^S.txt";
	int NRealPoles = 1;
	int NComplexPoles = 16;

	//###########################Reading File########################################

	

	//Ali: var to store freq points in data (upto 1024 data points)
	double *freq;
	hipMallocManaged(&freq, SIZE * sizeof(double));

	//Ali: store collected data in complex form (upto 1024 data points)
	hipComplex *data;
	hipMallocManaged(&data, SIZE * sizeof(hipComplex));

	//Ali: store info about the stroed data
	//		-lowest freq
	//		-highest freq
	//		-number of freq points
	frequency *frequencyInfo;
	hipMallocManaged(&frequencyInfo, sizeof(frequency));

	std::string::size_type sz;
	

	ifstream infile(dataFileName);
	
	//Ali: Attempt to open data file 
	if (!infile) {
		std::cout << "While opening data file an error was encountered" << std::endl;
	} else {
		string line;
		int fileColumn, freqCount = 0;
		(*frequencyInfo).fpointCount = 0;
		bool skipline;
		string word;

		//Ali: iterate through each line
		while (getline(infile, line)) {
			istringstream stringOfLine(line);
			fileColumn = 0;

			//Ali: Iterrate through each element of line which is refered to as a column
			while (stringOfLine) {
				//load new element into a string variable called "word"
				stringOfLine >> word;
				skipline = false;

				//Ali: if comment ("%#") is detected then like is skipped
				if (word.compare("%#") == 0) {
					skipline = true;
					break;
				}

				//Ali: if it is the first column then data is stored as the frequency
				if (fileColumn == 0) {
					//Ali: translate string to int 
					freq[freqCount] = stod(word, &sz);

					//Ali: if it is the forst freq reading then initialize max and min
					if (freqCount == 0) {
						(*frequencyInfo).high = freq[freqCount];
						(*frequencyInfo).low = freq[freqCount];
					}else {
						//Ali: if current freq is greater than highest then update 
						if ((*frequencyInfo).high < freq[freqCount]) {
							(*frequencyInfo).high = freq[freqCount];
						}
						//Ali: if current freq is smaller than lowest then update 
						if ((*frequencyInfo).low > freq[freqCount]) {
							(*frequencyInfo).low = freq[freqCount];
						}	
					}

				}
				//Ali: The second column is the real part of the response
				if (fileColumn == 1) {
					data[freqCount].x = stod(word, &sz);
				}
				//Ali: The third column is the imag part of the response
				if (fileColumn == 2) {
					data[freqCount].y = stod(word, &sz);
				}
				fileColumn++;
			}
			//Ali: If line is skipped then dont add to freq count
			if (!skipline) {
				freqCount++;
			}		
		}

		//Ali: Update total freq count
		(*frequencyInfo).fpointCount= freqCount;

		//Ali: make sure the min is atleast 1e-6
		if ((*frequencyInfo).low < MINFREQ) {
			(*frequencyInfo).low = MINFREQ;
		}
	}

	if (debug) {
		printf("\n\n********************************************************\n");
		printf("extracted data\n");
		printf("********************************************************\n");
		for (int z = 0; z < (*frequencyInfo).fpointCount; z++) {
			printf("Z: %d FREQ: %f RESP: %f(%f) \n", z, freq[z], data[z].x, data[z].y);
		}
		printf("^^^^^^^^^^^^^^^^^^^\n");
		printf("HiegestFREQ: %f GHz \n LowestFREQ: %f GHz \n FreqPoints %d\n", (*frequencyInfo).high, (*frequencyInfo).low, (*frequencyInfo).fpointCount);
	}



	//###########################Initial Pole Guess########################################

	//This constant is predetermined in a paper
	int Real_part_Divisor = 100;

	int NumberOfPoles = NRealPoles + (NComplexPoles / 2);

	double *Poles_imag_part;
	hipMallocManaged(&Poles_imag_part, NumberOfPoles * sizeof(double));

	double *Poles_real_part;
	hipMallocManaged(&Poles_real_part, NumberOfPoles * sizeof(double));

	double *Real_Poles;
	hipMallocManaged(&Real_Poles, NRealPoles * sizeof(double));

	hipComplex *Complex_Poles;
	hipMallocManaged(&Complex_Poles, NComplexPoles * sizeof(hipComplex));

	int B[2] = { 1, 1 };
	int C[2] = { 1, -1 };

	double poleSpacing = ((*frequencyInfo).high - (*frequencyInfo).low) / (NumberOfPoles-1);

	for (int z = 0; z < NumberOfPoles; z++) {
		Poles_imag_part[z] = (*frequencyInfo).low + poleSpacing*z;
		Poles_real_part[z] = -Poles_imag_part[z] / Real_part_Divisor;
	}

	//Set Real Poles
	for (int z = 0; z < NRealPoles; z++) {
		Real_Poles[z] = 2*M_PI*Poles_real_part[z];
	}
	
	//Set Complex Poles
	int poleIndex = 0;
	for (int z = 0; z < NComplexPoles/2; z++) {
		for (int i = 0; i< 2; i++) {
			Complex_Poles[poleIndex].x = 2*M_PI*(Poles_real_part[NRealPoles + z] * B[i]);
			Complex_Poles[poleIndex].y = 2*M_PI*(Poles_imag_part[NRealPoles + z] * C[i]);
			poleIndex++;
		}
	}


	printf("\n\n********************************************************\n");
	printf("Initial Poles\n");
	printf("********************************************************\n");
	for (int z = 0; z < NRealPoles; z++) {
		printf("Real Pole[%d]: %f(%f) \n", z, Real_Poles[z]);
	}
	for (int z = 0; z <  NComplexPoles; z++) {
		printf("Complex Pole[%d]: %f(%f) \n", z, Complex_Poles[z].x, Complex_Poles[z].y);
	}
	printf("^^^^^^^^^^^^^\n");
	printf("NumberOfPoles: %d  \n poleSpacing: %f \n", NumberOfPoles, poleSpacing);

	//###########################Ahat set up########################################
	hipComplex *Ahat;
	//size_t pitch;
	//hipMallocPitch(&devPtr, &devPitch, Ncols * sizeof(float), Nrows);
	//hipMallocPitch(&Ahat, &pitch, Ncols * sizeof(float), Nrows));
	int NPorts = 1;
	int* Apattern;
	hipMallocManaged(&Apattern, (NRealPoles + NComplexPoles+ NPorts) * sizeof(int));

	int isReal = 1;
	for (int i = 0; i < NRealPoles + NComplexPoles * 2 + NPorts; i++) {
		if (i < NRealPoles) {
			Apattern[i]=0;	
		}else if (i < NRealPoles+NComplexPoles) {
			Apattern[i] = (isReal) ? 1 : -1;
			isReal ^= 1;	
		}else if (i < NRealPoles + NComplexPoles+NPorts) {
			Apattern[i] = 2;
		}
	};
	printf(" A Pattern: ");
	for (int i = 0; i < NRealPoles + NComplexPoles * 2 + NPorts; i++) {
			printf("%d ", Apattern[i]);
	};
	printf("\n");

	int NCol = NRealPoles * 2 + NComplexPoles * 2;
	int NRow = (*frequencyInfo).fpointCount;
	hipMallocManaged(&Ahat, NRow*((NRealPoles + NComplexPoles) * sizeof(double)));
	double s;
	printf("here 1\n");
	int g = 0;
/*	for (int col = 0; col < NRealPoles; col++) {
		for (int row = 0; row < (*frequencyInfo).fpointCount; row++) {
			s = 2 * M_PI*freq[row];
			//Ahat[col*NRow + row].x = (1 / (freq[row] - Real_Poles[col]);
			Ahat[col*NRow + row].x = -Real_Poles[col]/(pow(Real_Poles[col],2) + pow(s,2));
			Ahat[col*NRow + row].y = -s / (pow(Real_Poles[col], 2) + pow(s, 2));
			g++;
		};
	}; */

	printf("here 2\n");
	double real;
	double imag;
	int h= 0;
	int y;
	int test = 0;
	for (int col = 0; col < NComplexPoles+NRealPoles+NPorts; col++) {
		printf("%d   ", Apattern[col]);
		if (Apattern[col] == 0) {
			real = Real_Poles[col];
			imag = 0;
		}
		else {
			real = Complex_Poles[col-NRealPoles].x;
			imag = Complex_Poles[col- NRealPoles].y;
		}

		for (int row = 0; row < (*frequencyInfo).fpointCount; row++) {
			y = NRealPoles + col;


			s = 2 * M_PI*freq[row];
			
//real		
			if(Apattern[col] == 0){
				Ahat[col*NRow + row].x = -real / (pow(real, 2) + pow(s, 2));
				Ahat[col*NRow + row].y = -s / (pow(real, 2) + pow(s, 2));
			}else if (Apattern[col] == 1) {
				double denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
				Ahat[col*NRow + row].x = (-2 * real*(pow(real, 2) + pow(s, 2) + pow(imag, 2))) / denum;
				Ahat[col*NRow + row].y = (-2 * s *(pow(real, 2) + pow(s, 2) + pow(imag, 2))) / denum;
			}else if (Apattern[col] == -1) {
				double denum = (pow(real, 2)*(pow(real, 2) + 2 * pow(s, 2) + 2 * pow(imag, 2)) + pow(imag, 4) - 2 * pow(imag, 2)*pow(s, 2) + pow(s, 4));
				Ahat[(col)*NRow + row].x = (-2 * imag*(pow(real, 2) - pow(s, 2) + pow(imag, 2))) / denum;
				Ahat[(col)*NRow + row].y = (-4 * real*imag*s) / denum;
			}			
			//imag			
			g++;

			
		};
		h++;
	};
	printf("\n");
	for (int row = 0; row < 17; row++) {
		for (int col = 0; col < 7; col++) {
			printf(" %.4e(%f)", Ahat[col*NRow + row].x, Ahat[col*NRow + row].y);
		};
		printf("\n");
	};


/*	Poles_imag_part = linspace(f.L, f.H, IP.Nreal + IP.Ncomplex / 2);
	Poles_real_part = -Poles_imag_part / Real_part_Divisor;

	Real_Poles = Poles_real_part(1 : IP.Nreal);

	Complex_Poles = ...
		kron(Poles_real_part(IP.Nreal + 1:end), [1, 1]) + ...
		kron(Poles_imag_part(IP.Nreal + 1:end), [-j, j]);

	initial_Poles = 2 * pi*transpose(cat(2, Real_Poles, Complex_Poles)); */

	

	hipFree(freq);
	hipFree(data);
	hipFree(frequencyInfo);
	hipFree(Poles_imag_part);
	hipFree(Poles_real_part);
	hipFree(Real_Poles);
	hipFree(Complex_Poles);

	return 0;
}